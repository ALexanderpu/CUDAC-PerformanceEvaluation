#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <time.h>
#include <sys/time.h>
#include <random>
#include <bits/stdc++.h>
#include <iomanip>
using namespace std;

// #define DSIZE 1000;

static const double min_weight = 0.000001;

const double qnan = std::numeric_limits<double>::quiet_NaN();

pair<vector<double>, vector<double> > parse_csv(std::string &csvfile){
    std::ifstream data(csvfile);
    std::string line;
    std::vector<std::vector<double> > csvdata;
    unsigned long length = 0;
    while(getline(data, line)){
        std::stringstream lineStream(line);
        std::string cell;
        std::vector<double> parsedRow;
        while(getline(lineStream,cell,',')) //include head
        {
            parsedRow.push_back(strtof(cell.c_str(), 0));
        }
        length += 1;
        csvdata.push_back(parsedRow);
    }
    vector<double> x, y;
    cout << length << " size "<< endl;
    for(int i = 1; i < length; i++){
        x.push_back(csvdata[i][1]);
        y.push_back(csvdata[i][2]);
    }
    return make_pair(x, y);
}


double dist_func(const std::vector<double>& A, const std::vector<double>& B){
    double dist = 0;
    for (auto a_iter = A.begin(), b_iter = B.begin(); a_iter != A.end(); ++a_iter, ++b_iter)
        dist += (*a_iter - *b_iter) * (*a_iter - *b_iter);

    return sqrt(dist);
}


template<typename T>
vector<size_t> index_sort(const vector<T>& v){
	vector<size_t> result(v.size());
	iota(begin(result), end(result), 0);
	sort(begin(result), end(result), [&v](const double &lhs, const double &rhs){return v[lhs] < v[rhs];});
	return result;
}

int main(int argc, char *argv[])
{
    timeval t1, t2;
    int num_gpus = 0;   // number of CUDA GPUs
    int num_cpus = 0;

    printf("%s Starting...\n\n", argv[0]);

    // determine the number of CUDA capable GPUs
    hipGetDeviceCount(&num_gpus);
    // determine the number of cpu
    num_cpus = omp_get_num_procs();

    if (num_gpus < 1){
        cout << "no CUDA capable devices were detected" << endl;
        return 1;
    }

    // display CPU and GPU configuration
    printf("number of host CPUs:\t%d\n", num_cpus);
    printf("number of CUDA devices:\t%d\n", num_gpus);

    for (int i = 0; i < num_gpus; i++)
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
        printf("   %d: %s\n", i, dprop.name);
    }

    typedef thrust::device_vector<int> dvec;
    typedef dvec *p_dvec;
    std::vector<p_dvec> dvecs;
    /*
    for(unsigned int i = 0; i < num_gpus; i++) {
      hipSetDevice(i);
      p_dvec temp = new dvec(DSIZE);
      dvecs.push_back(temp);
    }
    */

    // access the matrix
    vector<double > observations;
    vector<double > targets;
    string csvfile = "/home/bo/Documents/CCM-Parralization/TestCSVData/test_float_10000.csv";
    std::tie(observations, targets) = parse_csv(csvfile);
    //observations = {3, 4, 5, 6, 7, 1, 2, 5, 2};
    //targets = {7, 5, 8, 1, 3, 4, 3, 2, 1};


    cout << "print observations" << endl;
    for(auto ele: observations) cout << ele << " ";
    cout << endl;
    /*
    cout << "print targets" << endl;
    for(auto ele: targets) cout << ele << " ";
    cout << endl;
	*/



    if(observations.size() != targets.size()){
    	cout << "input sequence length not match" << endl;
    	return 1;
    }
    size_t num_vectors = observations.size();
    int num_samples = 250;
    size_t E = 3;
    size_t tau = 1;
    size_t lib_size = min((size_t)300, num_vectors);
    bool enable_cpus = true;
    bool enable_global_sort = true;
    bool enable_gpu = true;
    bool replacement = true;

    vector<vector<double>> lag_vector(num_vectors, vector<double>(E, qnan));
    // make lag vector
    for (size_t i = 0; i < (E - 1) * tau; ++i)
           for (size_t j = 0; j < E; ++j)
               if (i >= j * tau)
            	   lag_vector[i][j] = observations[i - j * tau];

    for(size_t i = (E-1)*tau; i < num_vectors; i++){
    	for(size_t j = 0; j < E; j++){
    		lag_vector[i][j] = observations[i - j*tau];
    	}
    }

    // print lag  vector
    for(size_t j = 0; j < E; j++){
    	for(size_t i = 0; i < num_vectors; i++){
    		cout << lag_vector[i][j] << " ";
    	}
    	cout << endl;
    }

    // specify index array
    cout << "specify index array: " << endl;
    vector<size_t> which_lib;
    vector<size_t> which_pred;
    size_t start_of_range = std::min((E - 1) * tau, num_vectors - 1);
    size_t end_of_range = num_vectors - 1;
    for (size_t j = start_of_range; j <= end_of_range; ++j){
    	which_lib.push_back(j);
        which_pred.push_back(j);
    }
    lib_size = min(which_lib.size(), lib_size);

    for(size_t i = 0; i < which_lib.size(); i++)cout << which_lib[i] << " ";
    cout << endl;
    for(size_t i = 0; i < which_pred.size(); i++)cout << which_pred[i] << " ";
    cout << endl;



    // compute distance matrix using lag_vector  N*E  (contain nan)
    cout << "calculate the distance matrix: " << endl;
    vector<vector<double> > distance_matrix(num_vectors, vector<double>(num_vectors,  std::numeric_limits<double>::max()));
    for(auto& cur_pred: which_pred){
    	for(auto& cur_lib: which_lib){
    		distance_matrix[cur_pred][cur_lib] = dist_func(lag_vector[cur_pred], lag_vector[cur_lib]);
    		distance_matrix[cur_lib][cur_pred] = distance_matrix[cur_pred][cur_lib];
    	}
    }
    /*
    // print the distance matrix
	cout << "distance matrix: " << endl;
    for(size_t i = 0; i < distance_matrix.size(); i++){
		for(size_t j = 0; j < distance_matrix[0].size(); j++){
			cout << distance_matrix[i][j] << " ";
		}
		cout << endl;
	}
	*/


    //TODO:  then globally sort here and search  - purpose: mitigating bottlenecks in multi-node computing   cpu; network utilization
    // test rank_matrix here
    vector<vector<size_t> > rank_matrix;
    if(enable_global_sort){

    	rank_matrix(num_vectors, vector<size_t>(num_vectors, 0));
		if(enable_gpu){
			gettimeofday(&t1,NULL);
			// gpu sort here
			for(auto& cur_pred: which_pred){
				thrust::device_vector<double> values_gpu(distance_matrix[cur_pred]);
				thrust::device_vector<size_t> indices_gpu(distance_matrix[cur_pred].size());
				thrust::sequence(indices_gpu.begin(), indices_gpu.end());
				thrust::sort_by_key(values_gpu.begin(), values_gpu.end(), indices_gpu.begin()); // this function will change values and indices at the same time
				thrust::copy(indices_gpu.begin(), indices_gpu.end(), rank_matrix[cur_pred].begin());  // copy device to host
			}

			gettimeofday(&t2,NULL);
			unsigned long et_gpu = ((t2.tv_sec * 1000000)+t2.tv_usec) - ((t1.tv_sec * 1000000) + t1.tv_usec);
			printf("gpu sorting running time = %fs\n", (float)et_gpu/(float)(1000000));

		}else{
		//if(enable_gpu){
			gettimeofday(&t1,NULL);
			for(auto& cur_pred: which_pred){
				vector<double>& values_cpu = distance_matrix[cur_pred];
				vector<size_t> indices_cpu(values_cpu.size());
				size_t n = 0;
				std::generate(indices_cpu.begin(), indices_cpu.end(), [&n]{return n++;});
				std::sort(indices_cpu.begin(), indices_cpu.end(), [&values_cpu](size_t i1, size_t i2){return values_cpu[i1] < values_cpu[i2];});
				std::copy(indices_cpu.begin(), indices_cpu.end(), rank_matrix[cur_pred].begin());
			}
			gettimeofday(&t2,NULL);
			unsigned long et_cpu = ((t2.tv_sec * 1000000)+t2.tv_usec) - ((t1.tv_sec * 1000000) + t1.tv_usec);
			printf("cpu sorting running time = %fs\n", (float)et_cpu/(float)(1000000));
		}
		/*
		// print the rank matrix
		cout << "rank matrix: " << endl;
		for(size_t i = 0; i < rank_matrix.size(); i++){
			for(size_t j = 0; j < rank_matrix[0].size(); j++){
				cout << rank_matrix[i][j] << " ";
			}
			cout << endl;
		}
		*/
    }

	gettimeofday(&t1,NULL);
    size_t seed = (size_t)(std::chrono::high_resolution_clock::now().time_since_epoch().count());
    std::mt19937 rng(seed);
	std::uniform_int_distribution<uint32_t> lib_sampler(0, (unsigned int)(which_lib.size() - 1));
	std::uniform_real_distribution<double> unif_01(0, 1);
	size_t max_lib_size = which_lib.size();


    gettimeofday(&t2,NULL);
	unsigned long et1 = ((t2.tv_sec * 1000000)+t2.tv_usec) - ((t1.tv_sec * 1000000) + t1.tv_usec);
	printf("stage 1 running time = %fs\n", (float)et1/(float)(1000000));


	gettimeofday(&t1,NULL);

	if(enable_cpus){
		omp_set_num_threads(num_cpus);
	}else{
		omp_set_num_threads(1);
	}
	// final result
	vector<double > rhos;

    #pragma omp parallel for
	for(size_t sample = 0; sample < num_samples; sample++){
		unsigned int cpu_thread_id = omp_get_thread_num();
		// cout << "initialize data for thread id: " << cpu_thread_id << " process: " << sample<< endl;

		// sample l size of observation index
		vector<size_t> lib_contain_indices;

		if(replacement){
			// sample with replacement (default)
			for(auto l = 0; l < lib_size; l++)
				lib_contain_indices.push_back(which_lib[lib_sampler(rng)]);
		}else{
			// sample without replacement   (refer from the algorithm from Knuth)
			lib_contain_indices.assign(lib_size, 0);
			size_t m = 0;
			size_t t= 0;
			while(m < lib_size){
				if(double(max_lib_size - t) * unif_01(rng) >= double(lib_size - m)){
					++t;
				}
				else{
					lib_contain_indices[m] = which_lib[t];
					++t; ++m;
				}
			}
		}
		// find nearest neighbor here?


		// initialize predicted
		vector<double> predicted(num_vectors, qnan);
		// simplex prediction to compute predicted
		size_t cur_pred_index, num_ties;
		std::vector<double > weights;
		for(size_t k = 0; k < which_pred.size(); k++){
			cur_pred_index = which_pred[k];

			vector<size_t> lib;
			// TODO: filter the index itself
			std::copy_if(lib_contain_indices.begin(), lib_contain_indices.end(), back_inserter(lib), [&cur_pred_index](size_t i){return i != cur_pred_index;});
			// cout << lib.size() << endl;

			if(!enable_global_sort){
				//find nearest neighbors without global sorted table here: distance_matrix[cur_pred_index]
				std::vector<size_t> neighbors;
				const vector<double>& distances = distance_matrix[cur_pred_index];
				std::sort(lib.begin(), lib.end(), [&distances](size_t i1, size_t i2){return distances[i1] < distances[i2];});
			}else{
				//find nearest neighbors with global sorted table
				// when lib size is large


			}


			// identify tie
			size_t tie_index = min(lib.size()-1, E);
			double tie_distance = distance_matrix[cur_pred_index][lib[tie_index]];
			size_t cur_tie_index = tie_index;
			for(; cur_tie_index < lib.size(); cur_tie_index++){
				if(distance_matrix[cur_pred_index][lib[cur_tie_index]] > tie_distance){
					cur_tie_index -= 1; // is the previous one
					break;
				}
			}
			// 0 - cur_tie_index   in lib   is the neighbor index range

			double min_distance = distance_matrix[cur_pred_index][lib[0]];
			weights.assign(cur_tie_index+1, min_weight);
			for(size_t i = 0; i < cur_tie_index; i++){
				if(distance_matrix[cur_pred_index][lib[i]] == 0){
					cout << "this is special case "<< endl;
					weights[i] = 1;
				}else if(min_distance != 0){
					weights[i] = fmax(exp(-distance_matrix[cur_pred_index][lib[i]] / min_distance), min_weight);
				}
			}

			// identify tie exist and adjust weights
			if(cur_tie_index > tie_index){
				num_ties = 0;
				int left_tie = tie_index-1;
				while(left_tie >= 0 && distance_matrix[cur_pred_index][lib[left_tie]] == tie_distance){
					left_tie--;
					num_ties++;
				}
				int right_tie = tie_index+1;
				while(right_tie <= cur_tie_index && distance_matrix[cur_pred_index][lib[right_tie]] == tie_distance){
					right_tie--;
					num_ties++;
				}
				double tie_adj_factor = double(num_ties  - cur_tie_index + tie_index) / double(num_ties);

				for(size_t t = 0; t <= cur_tie_index; t++){
					if(distance_matrix[cur_pred_index][lib[t]] == tie_distance)
						weights[t] *= tie_adj_factor;
				}
			}

			// make prediction
			double total_weight = accumulate(weights.begin(), weights.end(), 0.0);
			predicted[cur_pred_index] = 0;
			for(size_t t = 0; t <= cur_tie_index; t++){
				predicted[cur_pred_index] += weights[t] * targets[lib[t]];
			}
			// normalized
			predicted[cur_pred_index] = predicted[cur_pred_index] / total_weight;
		}

		// compute rho for every sample between predicted and targets
		size_t num_pred = 0;
		double sum_tar = 0;
		double sum_pred = 0;
		double sum_squared_tar = 0;
		double sum_squared_pred = 0;
		double sum_prod = 0;
		if(targets.size() == predicted.size()){
			for(size_t k = 0; k < targets.size(); k++){
				if(!std::isnan(predicted[k]) && !std::isnan(targets[k])){
					num_pred += 1;
					sum_tar += targets[k];
					sum_pred += predicted[k];
					sum_squared_tar += targets[k] * targets[k];
					sum_squared_pred += predicted[k] * predicted[k];
					sum_prod += targets[k] * predicted[k];
				}
			}
		}
		double rho = 0;
		double denominator = sqrt((sum_squared_tar * num_pred - sum_tar * sum_tar) * (sum_squared_pred * num_pred - sum_pred * sum_pred));
		double numerator = (sum_prod * num_pred - sum_tar * sum_pred);
		if(denominator != 0)
			rho = numerator / denominator;

		#pragma omp critical
		rhos.push_back(rho);
	 }




        /*
        thrust::host_vector<int> data(DSIZE);
        thrust::generate(data.begin(), data.end(), rand);

        // copy data
        // critical part: should enter into task
        for (unsigned int i = 0; i < num_gpus; i++) {
            hipSetDevice(i);
            thrust::copy(data.begin(), data.end(), (*(dvecs[i])).begin());
          }

        printf("start sort\n");


        hipSetDevice(cpu_thread_id);
        thrust::sort((*(dvecs[cpu_thread_id])).begin(), (*(dvecs[cpu_thread_id])).end());
        hipDeviceSynchronize();
		*/

	cout << " the result size: " << rhos.size() << endl;
	for(size_t i = 0; i < rhos.size(); i++){
		cout << rhos[i] << " ";
	}
	cout << endl;

    gettimeofday(&t2,NULL);
    unsigned long et2 = ((t2.tv_sec * 1000000)+t2.tv_usec) - ((t1.tv_sec * 1000000) + t1.tv_usec);
    printf("stage 2 runing time = %fs\n", (float)et2/(float)(1000000));


    printf("total runing time = %fs\n", (float)(et1+et2)/(float)(1000000));

    /*
    unsigned long et = ((t2.tv_sec * 1000000)+t2.tv_usec) - ((t1.tv_sec * 1000000) + t1.tv_usec);
    if (hipSuccess != hipGetLastError())
        printf("%s\n", hipGetErrorString(hipGetLastError()));
    printf("sort time = %fs\n", (float)et/(float)(1000000));
    // check results
    thrust::host_vector<int> result(DSIZE);
    for (int i = 0; i < num_gpus; i++)
    {
        hipSetDevice(i);
        thrust::copy((*(dvecs[i])).begin(), (*(dvecs[i])).end(), result.begin());
        for (int j = 0; j < DSIZE; j++)
          if (data[j] != result[j]) { printf("mismatch on device %d at index %d, host: %d, device: %d\n", i, j, data[j], result[j]); return 1;}
    }
    */
    printf("Success\n");
    return 0;
}
